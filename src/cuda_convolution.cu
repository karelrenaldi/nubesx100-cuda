#include "hip/hip_runtime.h"
% % writefile test.cu

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define DATAMAX 1000
#define DATAMIN -1000
#define BLOCK_SIZE 1024

    void
    input_matrix(int *matrix, int num_elements)
{
    for (int i = 0; i < num_elements; i++)
    {
        scanf("%d", &matrix[i]);
    }
}

void print_matrix(int *matrix, int num_elements)
{
    for (int i = 0; i < num_elements; i++)
    {
        printf("%i: %d\n", i, matrix[i]);
    }
}

int ceil_division(int a, int b)
{
    return a / b + (a % b != 0);
}

__device__ void change_idx_1d_to_2d(int idx, int ncol, int *row, int *col)
{
    *row = idx / ncol;
    *col = idx % ncol;
}

__device__ int change_idx_2d_to_1d(int ncol, int row, int col)
{
    return ncol * row + col;
}

__global__ void convolution(
    int *d_output,
    int *d_matrix,
    int *d_kernel,
    int output_row,
    int output_col,
    int matrix_row,
    int matrix_col,
    int kernel_row,
    int kernel_col)
{
    int output_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (output_idx >= output_row * output_col)
    {
        return;
    }

    int offset = blockIdx.y * matrix_row * matrix_col;

    int output_i, output_j;
    change_idx_1d_to_2d(output_idx, output_col, &output_i, &output_j);

    int res = 0;
    int curr_kernel_idx, curr_matrix_idx;
    for (int kernel_i = 0; kernel_i < kernel_row; kernel_i++)
    {
        for (int kernel_j = 0; kernel_j < kernel_col; kernel_j++)
        {
            curr_kernel_idx = change_idx_2d_to_1d(kernel_col, kernel_i, kernel_j);
            curr_matrix_idx = change_idx_2d_to_1d(
                                  matrix_col,
                                  kernel_i + output_i,
                                  kernel_j + output_j) +
                              offset;

            res += d_kernel[curr_kernel_idx] * d_matrix[curr_matrix_idx];
        }
    }

    d_output[output_idx + (blockIdx.y * output_row * output_col)] = res;
}

__global__ void find_range(
    int *d_range_output,
    int *d_conv_input,
    int conv_row,
    int conv_col,
    int num_output,
    int datamax,
    int datamin)
{
    int output_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (output_idx >= num_output)
    {
        return;
    }

    int curr_max = datamin, curr_min = datamax;
    int num_conv_elements = conv_row * conv_col;
    int offset = (blockIdx.x * blockDim.x + threadIdx.x) * num_conv_elements;

    for (int i = offset; i < num_conv_elements + offset; i++)
    {
        int curr_el = d_conv_input[i];
        if (curr_el > curr_max)
        {
            curr_max = curr_el;
        }
        if (curr_el < curr_min)
        {
            curr_min = curr_el;
        }
    }

    d_range_output[output_idx] = curr_max - curr_min;
}

int main()
{
    int kernel_row, kernel_col;
    scanf("%d %d", &kernel_row, &kernel_col);

    int num_kernel_elements = kernel_row * kernel_col;
    int kernel_size = num_kernel_elements * sizeof(int);

    int *kernel, *d_kernel;
    kernel = (int *)malloc(kernel_size);
    hipMalloc((void **)&d_kernel, kernel_size);

    input_matrix(kernel, num_kernel_elements);
    hipError_t errKernel = hipMemcpy(d_kernel, kernel, kernel_size, hipMemcpyHostToDevice);
    if (errKernel != hipSuccess)
    {
        printf("Failed to copy kernel to GPU\n");
    }

    int matrix_row, matrix_col, num_matrix;
    scanf("%d %d %d", &num_matrix, &matrix_row, &matrix_col);

    int num_matrix_elements = num_matrix * matrix_row * matrix_col;
    int matrix_size = num_matrix_elements * sizeof(int);

    int *matrix, *d_matrix;
    matrix = (int *)malloc(matrix_size);
    hipMalloc((void **)&d_matrix, matrix_size);

    input_matrix(matrix, num_matrix_elements);
    hipError_t errMatrix = hipMemcpy(d_matrix, matrix, matrix_size, hipMemcpyHostToDevice);
    if (errMatrix != hipSuccess)
    {
        printf("Failed to copy matrix to GPU\n");
    }

    int conv_output_row = matrix_row - kernel_row + 1;
    int conv_output_col = matrix_col - kernel_col + 1;
    int num_output_elements = num_matrix * conv_output_row * conv_output_col;
    int conv_output_size = num_output_elements * sizeof(int);

    int *conv_output, *d_output;
    conv_output = (int *)malloc(conv_output_size);
    hipMalloc((void **)&d_output, conv_output_size);

    dim3 block_dim(BLOCK_SIZE);
    dim3 grid_dim(ceil_division(num_output_elements, BLOCK_SIZE), num_matrix);

    convolution<<<grid_dim, block_dim>>>(
        d_output,
        d_matrix,
        d_kernel,
        conv_output_row,
        conv_output_col,
        matrix_row,
        matrix_col,
        kernel_row,
        kernel_col);

    // Get convolution output from GPU.
    hipMemcpy(conv_output, d_output, conv_output_size, hipMemcpyDeviceToHost);

    int *d_conv_input;
    hipMalloc((void **)&d_conv_input, conv_output_size);
    hipError_t errConv = hipMemcpy(d_conv_input, conv_output, conv_output_size, hipMemcpyHostToDevice);
    if (errConv != hipSuccess)
    {
        printf("Failed to copy convolution result to GPU\n");
    }

    int *range_output, *d_range_output;
    int range_output_size = num_matrix * sizeof(int);
    range_output = (int *)malloc(range_output_size);
    hipMalloc((void **)&d_range_output, range_output_size);

    find_range<<<ceil_division(num_matrix, BLOCK_SIZE), block_dim>>>(
        d_range_output,
        d_conv_input,
        conv_output_row,
        conv_output_col,
        num_matrix,
        DATAMAX,
        DATAMIN);

    // Get range output from GPU.
    hipMemcpy(range_output, d_range_output, range_output_size, hipMemcpyDeviceToHost);
    print_matrix(range_output, num_matrix);

    // Cleanup.
    hipFree(d_kernel);
    hipFree(d_matrix);
    hipFree(d_output);
    hipFree(d_conv_input);
    hipFree(d_range_output);
    free(kernel);
    free(matrix);
    free(conv_output);
    free(range_output);
}